/**
 *
 * OHIO STATE UNIVERSITY SOFTWARE DISTRIBUTION LICENSE
 *
 * Load-balanced sparse MTTKRP on GPUs (the “Software”) Copyright (c) 2019, The Ohio State
 * University. All rights reserved.
 *
 * The Software is available for download and use subject to the terms and
 * conditions of this License. Access or use of the Software constitutes acceptance
 * and agreement to the terms and conditions of this License. Redistribution and
 * use of the Software in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the capitalized paragraph below.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the capitalized paragraph below in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. The names of Ohio State University, or its faculty, staff or students may not
 * be used to endorse or promote products derived from the Software without
 * specific prior written permission.
 *
 * THIS SOFTWARE HAS BEEN APPROVED FOR PUBLIC RELEASE, UNLIMITED DISTRIBUTION. THE
 * SOFTWARE IS PROVIDED “AS IS” AND WITHOUT ANY EXPRESS, IMPLIED OR STATUTORY
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF ACCURACY, COMPLETENESS,
 * NONINFRINGEMENT, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED.  ACCESS OR USE OF THE SOFTWARE IS ENTIRELY AT THE USER’S RISK.  IN
 * NO EVENT SHALL OHIO STATE UNIVERSITY OR ITS FACULTY, STAFF OR STUDENTS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  THE SOFTWARE
 * USER SHALL INDEMNIFY, DEFEND AND HOLD HARMLESS OHIO STATE UNIVERSITY AND ITS
 * FACULTY, STAFF AND STUDENTS FROM ANY AND ALL CLAIMS, ACTIONS, DAMAGES, LOSSES,
 * LIABILITIES, COSTS AND EXPENSES, INCLUDING ATTORNEYS’ FEES AND COURT COSTS,
 * DIRECTLY OR INDIRECTLY ARISING OUT OF OR IN CONNECTION WITH ACCESS OR USE OF THE
 * SOFTWARE.
 *
 */

/**
 *
 * Author:
 *          Israt Nisa (nisa.1@osu.edu)
 *
 * Contacts:
 *          Israt Nisa (nisa.1@osu.edu)
 *          Jiajia Li (jiajia.li@pnnl.gov)
 *          Aravind Sukumaran-Rajam (sukumaranrajam.1@osu.edu)
 *          P.(Saday) Sadayappan (sadayappan.1@osu.edu)
 *
 */

#include <fstream>
#include <stdio.h>
#include <algorithm>
#include <iterator>
#include <utility>  
#include <math.h> 
#include <omp.h>
#include <hip/hip_runtime.h>
#include "mttkrp_cpu.h"
#include "mttkrp_gpu.h" 
#include <bits/stdc++.h>  

using namespace std;

int main(int argc, char* argv[]){ 
 
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    Options Opt = parse_cmd_options(argc, argv);

    Tensor X;
    load_tensor(X, Opt);
    sort_COOtensor(X);
    
    TiledTensor TiledX[Opt.nTile];
      
    Matrix *U = new Matrix[X.ndims]; 
    create_mats(X, U, Opt, false);
    randomize_mats(X, U, Opt);
    zero_mat(X, U, Opt.mode);

    if(Opt.verbose)
        cout << endl << "Starting MTTKRP..." << endl;  
    
    // print tensors and statistics
    if(Opt.impType == 0){
        double t0 = seconds();
        // print_COOtensor(X);
        create_HCSR(X, Opt);
        tensor_stats(X);
        // ((X.ndims == 3) ? print_HCSRtensor(X) : print_HCSRtensor_4D(X));  
    }
    // COO CPU   
    if(Opt.impType == 1){
        double t0 = seconds();
        ((X.ndims == 3) ?  MTTKRP_COO_CPU(X, U, Opt) :  MTTKRP_COO_CPU_4D(X, U, Opt));   
        printf("COO CPU - time: %.3f sec \n", seconds() - t0);
    }

    // HCSR CPU   
    else if(Opt.impType == 2){
        
        create_HCSR(X, Opt); 

        int *curMode = new int [X.ndims];
    
        for (int m = 0; m < X.ndims; ++m)
            curMode[m] = (m + Opt.mode) % X.ndims; 

        double t0 = seconds();        
        ((X.ndims == 3) ?  MTTKRP_HCSR_CPU(X, TiledX, U, Opt) :  MTTKRP_HCSR_CPU_4D(X, U, Opt)); 
        printf("gcc no opt : HCSR CPU - time: %.3f sec \n", seconds() - t0);        
    }

    // COO GPU  
    else if(Opt.impType == 3){
        MTTKRP_COO_GPU(X, U, Opt);
    }

    // HCSR GPU  
    else if(Opt.impType == 4){
        
        create_HCSR(X, Opt);
        MTTKRP_HCSR_GPU(X, U, Opt);
    }
    // HYB CPU
    else if(Opt.impType == 10){
        double t0 = seconds();
        create_HCSR(X, Opt);
        printf("create CSF - time: %.3f sec \n", seconds() - t0);
        HYBTensor HybX(X);
        cout << "Creating HYB... " ;
        t0 = seconds();
        ((X.ndims == 3) ?  create_HYB(HybX, X, Opt) :  create_HYB_4D(HybX, X, Opt));   
        printf("create HYB - time: %.3f sec \n", seconds() - t0);
        t0 = seconds();
        make_HybBin(HybX, Opt);
        printf("create bin - time: %.3f sec \n", seconds() - t0);
        // print_HYBtensor(HybX);      
        // ((X.ndims == 3) ?  MTTKRP_HYB_CPU(HybX, U, Opt) :  MTTKRP_HYB_CPU_4D(HybX, U, Opt));   
        MTTKRP_HYB_GPU(HybX, U, Opt);      
    }

    /* Tiled versions */
    else if(Opt.impType >= 5 && Opt.impType < 10){

        create_HCSR(X, Opt);

        int tilingMode = X.modeOrder[X.ndims -1];

        // make tile fit in shared
        if(Opt.impType == 9){
            Opt.tileSize = 192;
            Opt.nTile = (X.dims[tilingMode] + Opt.tileSize - 1)/Opt.tileSize;
        }
        else 
            Opt.tileSize = (X.dims[tilingMode] + Opt.nTile - 1)/Opt.nTile;  
        
        if(Opt.nTile > X.dims[tilingMode]){
            cout << "Number of tiles ("<< Opt.nTile << ") should be as minimum as K's dimension (" << X.dims[tilingMode]  << "). Exiting."<< endl ;
            exit(0);
        }

        // split X into tiles based on K indices
        make_KTiling(X, TiledX, Opt);
        
        // create HCSR for each tile
        for (int tile = 0; tile < Opt.nTile; ++tile){

            if(TiledX[tile].totNnz > 0){
                create_TiledHCSR(TiledX, Opt, tile);
            }
            // print_TiledHCSRtensor(TiledX, tile);
        }  

        // Split tiles into bins accordin to nnz in slice
        for (int tile = 0; tile < Opt.nTile; ++tile){
            if(TiledX[tile].totNnz > 0)
                make_TiledBin(TiledX, Opt, tile);
        }

        // COO GPU  
        if(Opt.impType == 5){
            double t0 = seconds();
            MTTKRP_TILED_COO_CPU(TiledX, U, Opt); 
            printf("TILED COO CPU - time: %.3f sec \n", seconds() - t0);  
        }

         // HCSR GPU  
        else if(Opt.impType == 6){
            double t0 = seconds();
            ((X.ndims == 3) ? MTTKRP_TILED_HCSR_CPU(TiledX, U, Opt) : MTTKRP_TILED_HCSR_CPU_4D(TiledX, U, Opt)); 
            printf("TILED HCSR CPU - time: %.3f sec \n", seconds() - t0); 
        }  

        // TILED COO GPU 
        else if(Opt.impType == 7){
            cout << "GPU COO has bugs! " << endl;
            MTTKRP_TILED_COO_GPU(TiledX, U, Opt);
        }

        // TILED HCSR GPU
        else if(Opt.impType == 8){
            cout << "Sorted mode: " << X.modeOrder[0] << " " << X.modeOrder[1] << " " <<X.modeOrder[2] << endl;
            MTTKRP_B_HCSR_GPU(TiledX, U, Opt);
        }
    }


    else // e.g. -1 
        cout << "no MTTKRP" << endl;

    if(!Opt.outFileName.empty()){
        write_output(U, Opt.mode, Opt.outFileName);
    }

    if(Opt.correctness){
        if (Opt.impType == 1) {
            cout << "Already running COO seq on CPU!" << endl; 
            exit(0);
        }
        if(Opt.verbose && Opt.impType == 12)
            cout << "checking only the last mode" << endl;
        
        int mode = Opt.mode;
        int nr = U[mode].nRows;  
        int nc = U[mode].nCols;
        DTYPE *out = (DTYPE*)malloc(nr * nc * sizeof(DTYPE));
        memcpy(out, U[mode].vals, nr*nc * sizeof(DTYPE));
        print_matrix(U, mode);

        // cout << "change double, mode sort, exec file" << endl; 

        randomize_mats(X, U, Opt);
        zero_mat(X, U, mode);

        cout << "correctness with COO on mode " << mode << endl;
        ((X.ndims == 3) ?  MTTKRP_COO_CPU(X, U, Opt) :  MTTKRP_COO_CPU_4D(X, U, Opt));
        print_matrix(U, mode);
        correctness_check(out, U[mode].vals, nr, nc);

    }
}


