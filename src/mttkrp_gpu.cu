#include "hip/hip_runtime.h"
/**
 *
 * OHIO STATE UNIVERSITY SOFTWARE DISTRIBUTION LICENSE
 *
 * Load-balanced sparse MTTKRP on GPUs (the “Software”) Copyright (c) 2019, The Ohio State
 * University. All rights reserved.
 *
 * The Software is available for download and use subject to the terms and
 * conditions of this License. Access or use of the Software constitutes acceptance
 * and agreement to the terms and conditions of this License. Redistribution and
 * use of the Software in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the capitalized paragraph below.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the capitalized paragraph below in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. The names of Ohio State University, or its faculty, staff or students may not
 * be used to endorse or promote products derived from the Software without
 * specific prior written permission.
 *
 * THIS SOFTWARE HAS BEEN APPROVED FOR PUBLIC RELEASE, UNLIMITED DISTRIBUTION. THE
 * SOFTWARE IS PROVIDED “AS IS” AND WITHOUT ANY EXPRESS, IMPLIED OR STATUTORY
 * WARRANTIES, INCLUDING, BUT NOT LIMITED TO, WARRANTIES OF ACCURACY, COMPLETENESS,
 * NONINFRINGEMENT, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED.  ACCESS OR USE OF THE SOFTWARE IS ENTIRELY AT THE USER’S RISK.  IN
 * NO EVENT SHALL OHIO STATE UNIVERSITY OR ITS FACULTY, STAFF OR STUDENTS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR
 * TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.  THE SOFTWARE
 * USER SHALL INDEMNIFY, DEFEND AND HOLD HARMLESS OHIO STATE UNIVERSITY AND ITS
 * FACULTY, STAFF AND STUDENTS FROM ANY AND ALL CLAIMS, ACTIONS, DAMAGES, LOSSES,
 * LIABILITIES, COSTS AND EXPENSES, INCLUDING ATTORNEYS’ FEES AND COURT COSTS,
 * DIRECTLY OR INDIRECTLY ARISING OUT OF OR IN CONNECTION WITH ACCESS OR USE OF THE
 * SOFTWARE.
 *
 */

/**
 *
 * Author:
 *          Israt Nisa (nisa.1@osu.edu)
 *
 * Contacts:
 *          Israt Nisa (nisa.1@osu.edu)
 *          Jiajia Li (jiajia.li@pnnl.gov)
 *          Aravind Sukumaran-Rajam (sukumaranrajam.1@osu.edu)
 *          P.(Saday) Sadayappan (sadayappan.1@osu.edu)
 *
 */

#include <iostream>
#include "mttkrp_gpu.h"
#include <vector>

inline hipError_t checkCuda(hipError_t result, int s){

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error in line : %s - %d\n", hipGetErrorString(result), s);
    assert(result == hipSuccess);
  }
  return result;
}

void cuda_timer_start(hipEvent_t start){
	checkCuda(hipEventRecord(start), __LINE__);
}
void cuda_timer_stop(hipEvent_t start, hipEvent_t stop, float &mili){
	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
}

// CUDA kernel call to do COO MTTKRP 
__global__ void mttkrp_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }  
		}
		__syncthreads();
	}
}
// CUDA kernel call to do COO MTTKRP 4D 
__global__ void mttkrp_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }
	    }  
	    __syncthreads();  
	}
}
//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_HYB_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }  
		}
		__syncthreads();
	}
}

//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
  ITYPE nnz,  DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] * dU3[idx3 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_HYB_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }
	    }  
	    __syncthreads();  
	}
}

__global__ void mttkrp_CSL_kernel(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = slc;//dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

__global__ void mttkrp_CSL_kernel_bin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

// CSL kernel with loop like ParTI
__global__ void mttkrp_CSL_kernel_bin_loop(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nSlices > nnz_per_loop) {
        num_loops_nnz = ((nSlices + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		slc = (gId + nl * nnz_per_loop) >> 5;
		              	              
		if(slc < nSlices){ 	    

			unsigned int mappedSlc = dSlcMapperBin[slc];
			unsigned int idx0 = dfbrIdx0[mappedSlc]; 
	    	int fb_st = fbrPtr0[mappedSlc];
			int fb_end = fbrPtr0[mappedSlc+1];
			tmp_val = 0;
			
			for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
				
			    unsigned int idx1 = dInds1[fbr];
		        unsigned int idx2 = dInds2[fbr];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
	            }   
			}
			for(unsigned int r=laneId; r<R; r+=32) {  
				atomicAdd(&dU0[idx0 * R + r], tmp_val);    
			}
		}
		__syncthreads();  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_CSL_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		tmp_val = 0;
		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		} 
	}
}

// HCSR MTTKRP : 16 WARP = 1 TB per slice
__global__ void mttkrp_HCSR_kernel_16WARP(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = tId >> 5; //(tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = blockIdx.x ;//gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0; 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        // unsigned int idx1 = dInds1[fbrPtr1[fbr]]; 
	        unsigned int idx1 = fbrIdx1[fbr];   
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}
// CUDA kernel call to do HCSR MTTKRP for the first bin 1 WARP per slice
__global__ void mttkrp_HCSR_kernel_COO(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int slc = gId >> 5; // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st; fbr < fb_end; fbr++){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	dU0[idx0 * R + r] += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// unsigned int slcPerTb = 16/warpPerSlice;
	// unsigned int shSlc = slc & slcPerTb;
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp);       
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }       
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;       
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], outbuffer);  
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1] ; fbr+=warpPerSlice){
			tmp_val = 0;
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];//dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	            // // atomicAdd(&dU0[idx0 * R + r], tmp);
	        }    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;

			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
            
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;  
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;     
		}
		for(unsigned int r=laneId; r<R; r+=32) { 
            atomicAdd(&dU0[idx0 * R + r], outbuffer);
        } 
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = (gId >> (5 + logOfWPC)) << 2; // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		bool diffFiber = false;
		unsigned int idx0;

		//maybe add another check fbr_fr can be larger than fiber?

		for (int fr = 0; fr < 4 && (fbr+fr) < (nFibers - 1); ++fr){

			diffFiber = false;
			unsigned int idx1 = fbrIdx1[fbr+fr];// dInds1[fbrPtr1[fbr]];  
			idx0 = fbrLikeSlcInds[fbr+fr];//slc;  
 			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr+fr] + workId; x < fbrPtr1[fbr+fr+1]; x+=warpPerSlice) {

		        unsigned int idx2 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; //2MR   
	            }       
	        }
	        	
        	for(unsigned int r=laneId; r<R; r+=32) { 
        		tmp += tmp_val * dU1[idx1 * R + r] ;
        	} 
	        
        	if(fbrLikeSlcInds[fbr+fr] != fbrLikeSlcInds[fbr+fr+1]) {

        		diffFiber = true;
	        	for(unsigned int r=laneId; r<R; r+=32) { 
	        		atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        	} 
        		tmp = 0;
        	}
        } 

        if(!diffFiber) {  
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }  
        }  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx2 = fbrLikeSlcInds[fbr];//slc;  
        
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx1 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
            }
        }     
        for(unsigned int r=laneId; r<R; r+=32) { 
        	tmp = tmp_val * dU2[idx2 * R + r] ;
        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
        }    
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx2 = fbrLikeSlcInds[fbr];//slc;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; x++) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; 
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); 
	        }    
		} 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// unsigned int slcPerTb = 16/warpPerSlice;
	// unsigned int shSlc = slc & slcPerTb;
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  

            // for(unsigned int r=laneId; r<R; r+=32) 
            // 	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	            	atomicAdd(&dU0[idx0 * R + r], tmp_val); //2MR
	                // atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            }
	        }   	
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  

		for(unsigned int r=laneId; r<R; r+=32) 
           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
        
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx0 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
                atomicAdd(&dU0[idx0 * R + r], tmp_val);
            }
        }         
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int warpId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //
	unsigned int blockId = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) //blockIdx.x ;//

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  

			for(unsigned int r=laneId; r<R; r+=32) 
	           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr] + warpId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
	                atomicAdd(&dU0[idx0 * R + r], tmp_val);
	            }
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]]; 

			for(unsigned int r=laneId; r<R; r+=32) 
            	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	// atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	                atomicAdd(&dU0[idx0 * R + r], tmp_val); 
	            }
	        }    
		} 
	}
}


int MTTKRP_COO_GPU(const Tensor &X, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2, *dInds3;
	DTYPE *dVals;

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds0, &(X.inds[mode0][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds1, &(X.inds[mode1][0]), X.totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;
    bool useLoop = true;
	
	// /* Like PARTI loop */ = 
	if(useLoop)
		grid.x = 32768;
	else 
		grid.x = (32 * X.totNnz + BLOCKSIZE - 1) / BLOCKSIZE;
	
	// CUDA call
	cuda_timer_start(start);

	if(!useLoop){

		if(X.ndims == 3)
			mttkrp_COO_kernel<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	// /* loop like ParTI */
	else{

		if(X.ndims == 3)
			mttkrp_COO_kernel_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R ); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	cuda_timer_stop(start, stop, mili);

	if(useLoop) cout << "Loop on. ";
    cout << "COO GPU using loop - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	// print_output(U, 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2); hipFree(dInds3);


	return 0;
}

int MTTKRP_HCSR_GPU(Tensor &X, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory
	cout << "FIX fiber idx" << endl;
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;
	DTYPE *dVals;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	// dummy bin mapper to be compatible with bin mapper when bin are not used
	X.slcMapperBin.push_back(std::vector<ITYPE>());      
	for (int s = 0; s < X.fbrIdx[0].size(); ++s)
		X.slcMapperBin[0].push_back(s);

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, X.slcMapperBin[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, X.fbrIdx[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, X.fbrPtr[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, X.fbrPtr[1].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, X.fbrIdx[1].size() * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dSlcMapperBin, &(X.slcMapperBin[0][0]), X.slcMapperBin[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr0, &(X.fbrPtr[0][0]), X.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx0, &(X.fbrIdx[0][0]), X.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr1, &(X.fbrPtr[1][0]), X.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx1, &(X.fbrIdx[1][0]), X.fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(X.ndims == 3){
		checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dFbrIdx2, X.fbrIdx[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, X.fbrPtr[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		
		checkCuda(hipMemcpy(dFbrPtr2, &(X.fbrPtr[2][0]), X.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrIdx2, &(X.fbrIdx[2][0]), X.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	grid.x = (Opt.warpPerSlice * 32 * X.dims[mode0] + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;

	checkCuda(hipEventRecord(start), __LINE__);

	// mttkrp_HCSR_kernel_COO<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
	// 	X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	if(X.ndims == 3)
		mttkrp_HCSR_kernel_smllBin<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	else
		mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds3, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		dFbrPtr2, dFbrIdx2, X.fbrIdx[0].size(), dU0, dU1, dU2, dU3, Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 


	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
    cout << "HCSR GPU - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);

	return 0;
}

int MTTKRP_TILED_COO_GPU(TiledTensor *TiledX, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2;
	ITYPE dLoc = 0, totNnz = 0;
	DTYPE *dVals;

	// All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];

	for (int tile = 0; tile < Opt.nTile; ++tile)
		totNnz += TiledX[tile].totNnz;

	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds0 + dLoc, &(TiledX[tile].inds[mode0][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds1 + dLoc, &(TiledX[tile].inds[mode1][0]), TiledX[tile].totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[mode2][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	// //Matrices
	DTYPE *dU0, *dU1, *dU2;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0, GPUTime = 0;

	// CUDA call
	dLoc = 0;
	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		cout << "Tile " << tile << " launched.. "<<endl;
		
		grid.x = (32 * TiledX[tile].totNnz + BLOCKSIZE - 1) / BLOCKSIZE;

		checkCuda(hipEventRecord(start), __LINE__);
		mttkrp_COO_kernel<<<grid, block>>>(dVals + dLoc, dInds0 + dLoc, dInds1 + dLoc, dInds2 + dLoc, TiledX[tile].totNnz, dU0, dU1, dU2,
								mode, R); 
	
		checkCuda(hipEventRecord(stop), __LINE__);
	    hipEventSynchronize(stop);
	    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
	    hipDeviceSynchronize();
	    cout << "Tile: " << tile << " - time " << mili << "ms"<< endl;
	    GPUTime += mili;
	   
	}
	cout << "COO GPU - time " << GPUTime << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2);

	return 0;
}

int MTTKRP_B_HCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){
	
	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];
    ITYPE mode3 =((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

	double t0 = seconds();
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(Opt.impType == 14)
			checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[tile].fbrLikeSlcInds[0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[mode3][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	float tnsMemcpyTime = seconds() - t0;

	t0 = seconds();
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	float mtxMemcpyTime = seconds() - t0;

	// cout << "tns and mtx memcopy time: " << tnsMemcpyTime <<", " << mtxMemcpyTime<< endl;
	
	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/
	int MTTKRPmode = mode0;//Opt.mode;
		
	for (int tile = 0; tile < Opt.nTile; ++tile){

		dBinLoc = 0;
		
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		BLOCKSIZE = 512;
		dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

		int smallBinEndsAt = 5;
		int slcPerTb = 0;

		double t0 = seconds();
		cuda_timer_start(start);
		
		// Process small bins.. accepts 2 slice 1 TB
		for (int bin = 0; bin < Opt.nBin ; ++bin){

			if(bin < smallBinEndsAt){
				
				ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

				dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

				grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

				if(TiledX[0].ndims == 3)
					mttkrp_HCSR_kernel_smllBin<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
					dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				else
					mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
					dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
			}
			
			// Processing heavy bin.. multiple TB per slice
			else{

				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
						
				grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
				
					if(TiledX[0].ndims == 3)
						mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
			}
		}
	
		cuda_timer_stop(start, stop, mili);
	    CPUtimer += seconds() - t0;
	    GPUTime += mili;

	    if(Opt.verbose){
	    	cout << "Tile: " << tile << " - time: " << mili << "ms";
	    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
	    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
			cout << endl;
		} 
	}
	
	allModeGPUTime += GPUTime;
	cout << "B-CSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_HYB_GPU(const HYBTensor &HybX, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE *dCOOInds0, *dCOOInds1, *dCOOInds2, *dCOOInds3;
	ITYPE *dCSLSlcPtr, *dCSLSlcInds, *dCSLInds1, *dCSLInds2, *dCSLSlcMapperBin;
	ITYPE *dfbrPtr0, *dfbrIdx0, *dInds2, *dInds3, *dfbrPtr1, *dfbrIdx1,  *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;

	DTYPE *dVals, *dCOOVals, *dCSLVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0, dFbrIdxLoc =0, dBinLoc = 0, dCSLBinLoc = 0, dFbrLoc2 =0;
	int warpPerSlice = Opt.warpPerSlice;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	// All tile same mode
	ITYPE mode0 = HybX.modeOrder[0];
    ITYPE mode1 = HybX.modeOrder[1];
    ITYPE mode2 = HybX.modeOrder[2];
    ITYPE mode3 =((HybX.ndims == 4) ? HybX.modeOrder[3] : 0) ;

    // ****** mem op HYB COO *******
    if(HybX.COOnnz > 0){
		
		checkCuda(hipMalloc((void**) &dCOOVals, HybX.COOnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds0, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds1, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds2, HybX.COOnnz * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCOOVals, &(HybX.COOvals[0]), HybX.COOnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds0, &(HybX.COOinds[mode0][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds1, &(HybX.COOinds[mode1][0]), HybX.COOnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds2, &(HybX.COOinds[mode2][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(HybX.ndims == 4){
			checkCuda(hipMalloc((void**) &dCOOInds3, HybX.COOnnz * sizeof(ITYPE)), 0);
			checkCuda(hipMemcpy(dCOOInds3, &(HybX.COOinds[mode3][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}

   // ****** mem op HYB CSL *******

	if(HybX.CSLnnz > 0){

		checkCuda(hipMalloc((void**) &dCSLVals, HybX.CSLnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcPtr,  HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcInds, HybX.CSLsliceIdx.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds1, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds2, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcMapperBin, HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCSLVals, &(HybX.CSLvals[0]), HybX.CSLnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);	
		checkCuda(hipMemcpy(dCSLSlcPtr + dSlcLoc, &(HybX.CSLslicePtr[0]), HybX.CSLslicePtr.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLSlcInds + dSlcIdxLoc, &(HybX.CSLsliceIdx[0]), HybX.CSLsliceIdx.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds1, &(HybX.CSLinds[mode1][0]), HybX.CSLnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds2, &(HybX.CSLinds[mode2][0]), HybX.CSLnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		dCSLBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dCSLBinLoc += HybX.CSLslcMapperBin[bin-1].size();

			if(HybX.CSLslcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, &(HybX.CSLslcMapperBin[bin][0]), HybX.CSLslcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
 
    // ****** mem op HYB HCSR *******

    if(HybX.HCSRnnz > 0){

		checkCuda(hipMalloc((void**) &dVals, HybX.HCSRnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr0,  HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx0, HybX.fbrIdx[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dSlcMapperBin, HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr1, HybX.fbrPtr[1].size()  * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx1, HybX.fbrPtr[1].size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dVals, &(HybX.vals[0]), HybX.HCSRnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0, &(HybX.fbrPtr[0][0]), HybX.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0, &(HybX.fbrIdx[0][0]), HybX.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1, &(HybX.fbrPtr[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1, &(HybX.fbrIdx[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	    if(HybX.ndims == 3){
	    	checkCuda(hipMalloc((void**) &dInds2, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	    	checkCuda(hipMemcpy(dInds2, &(HybX.inds[mode2][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

	    if(HybX.ndims == 4){
	        checkCuda(hipMalloc((void**) &dFbrIdx2, HybX.fbrIdx[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dFbrPtr2, HybX.fbrPtr[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dInds3, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	        checkCuda(hipMemcpy(dFbrPtr2, &(HybX.fbrPtr[2][0]), HybX.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2, &(HybX.fbrIdx[2][0]), HybX.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3, &(HybX.inds[mode3][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	    }

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += HybX.slcMapperBin[bin-1].size();

			if(HybX.slcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(HybX.slcMapperBin[bin][0]), HybX.slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(HybX.ndims == 4){
        checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
        checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    }
	
	// BLOCK and GRID
	int BLOCKSIZE = 512;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop, HYBstart, HYBstop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&HYBstart);
    hipEventCreate(&HYBstop);

    hipStream_t streams[2 * Opt.nBin + 1];
	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamCreate(&streams[bin]);

    float mili = 0, HYBmili =0, GPUTime = 0, CPUtimer = 0, HYBTime = 0;
	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0;
	bool useLoop = false;
	
	if(useLoop)
		grid.x = 32768*2;

			// mili = 0; 
	dCSLBinLoc = 0; dBinLoc = 0;

	int smallBinEndsAt = 5;
	int slcPerTb = 0;

	cuda_timer_start(HYBstart);

	// ******* CUDA COO *******

	// if(HybX.COOnnz > 0){

	// 	BLOCKSIZE = 128;
	// 	block.x = BLOCKSIZE;
	// 		// /* Like PARTI loop */ = 

	// 	if(!useLoop)
	// 		grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	// 	if(Opt.verbose) 
	// 		cuda_timer_start(start);
  		
 //  		if(!useLoop){

	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	// 	else{
  			
	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	//     if(Opt.verbose){
	//     	cuda_timer_stop(start, stop, mili);
	//     	HYBTime += mili;
	//     	cout << "HYB-COO GPU " << mili << "ms"<< endl;
	//     }
	// }
	// ******* CUDA CSL *******

	// if(HybX.CSLnnz > 0 || HybX.HCSRnnz > 0)
	{
		if(HybX.COOnnz > 0){

			BLOCKSIZE = 128;
			block.x = 128;
			grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	  		if(HybX.ndims == 3)
				mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
			else if (HybX.ndims == 4)
				mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
		
		}

		BLOCKSIZE = 512;
		block.x = BLOCKSIZE;

		for (int bin = 0; bin < Opt.nBin ; ++bin){

			dBinLoc += ((bin > 0) ? HybX.slcMapperBin[bin-1].size() : 0);
			dCSLBinLoc += ((bin > 0) ? HybX.CSLslcMapperBin[bin-1].size() : 0);

			if( HybX.slcMapperBin[bin].size() == 0 && HybX.CSLslcMapperBin[bin].size() == 0)
				continue;
			// Processing small bin.. merged to one. 1 WARP slice
			if(bin < smallBinEndsAt){

				warpPerSlice = 1;
				logOfWarpPerSlice = 0;//log2(warpPerSlice);
				slcPerTb = 16 / warpPerSlice;

				/* CSL small bin */
				if(HybX.CSLnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					mttkrp_CSL_kernel_bin<<<grid, block, 0, streams[1]>>>(dCSLVals, dCSLSlcInds, dCSLSlcMapperBin + dCSLBinLoc, 
						dCSLInds2, dCSLSlcPtr, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice); 
				}
				
				/* HCSR small bin */
				if(HybX.HCSRnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
					
					else if(HybX.ndims == 4)
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
				}
			}

			// Processing heavy bin.. multiple TB per slice
			else{
		
				TbPerSlc = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5

				if(TbPerSlc > 32) TbPerSlc = 32;		
				logOfTPS = log2(TbPerSlc);

				warpPerSlice = 16;
				logOfWarpPerSlice = 4;

				/* CSL big bin */
				if(HybX.CSLnnz > 0){	
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					mttkrp_CSL_kernel_hvyBin<<<grid, block, 0, streams[bin+1]>>>(dCSLVals + dLoc, dCSLSlcInds + dSlcIdxLoc, dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, 
						dCSLInds2 + dLoc, dCSLSlcPtr + dSlcLoc, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
				}

				/* HCSR big bin */
				if(HybX.HCSRnnz > 0){
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin+2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
							dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
						
					else if(HybX.ndims == 4)
	                    mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin + 2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
	                    dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
	                    dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS);
		        } 	

			}
		}

	    // if(Opt.verbose){
	    // 	cuda_timer_stop(start, stop, mili);
	    // 	HYBTime += mili;
	    // 	cout << "CSL+HCSR GPU-time: " << mili << "ms"<< endl;
	    // }
	}

	cuda_timer_stop(HYBstart, HYBstop, HYBmili);
	if(Opt.verbose)
		cout << "verbose on. HYB GPU: " << HYBmili << endl;
	else
		cout << "HYB GPU: " << HYBmili << endl;

	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamDestroy(streams[bin]);
	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); hipFree(dCOOVals); hipFree(dCSLVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
    hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
    hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dCSLInds1); hipFree(dCSLInds2); hipFree(dCSLSlcPtr); hipFree(dCSLSlcInds);
	hipFree(dCOOInds0); hipFree(dCOOInds1); hipFree(dCOOInds2); 

	return 0;
}